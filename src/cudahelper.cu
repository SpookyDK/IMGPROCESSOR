#include "cudahelper.h"
#include <hip/hip_runtime.h>

int getCudaDeviceCount() {
    int count = 0;
    hipGetDeviceCount(&count);
    return count;
}
